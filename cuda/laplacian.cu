#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cstring>
#include <IL/il.h>

__global__ void grayscale(unsigned char* data, unsigned char* out, int height, int width){
    auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
    auto y = (blockIdx.y * blockDim.y) + threadIdx.y;
    if(x >= width || y >= height) return;
    int i = (y*width)+x;
    int z = (height - y - 1) * width + x;
    out[z] = (307 * data[3 * i] + 604 * data[3 * i + 1] + 113 * data[3 * i + 2]) >> 10;
}

__global__ void edges(unsigned char* data, unsigned char* out, int height, int width, double strength){
    auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
    auto y = (blockIdx.y * blockDim.y) + threadIdx.y;
    if(x >= width || y >= height) return;
    int i = (y*width)+x;

    unsigned char result = data[i];

    if(x > 0 && x < width-1 && y > 0 && y < height-1)
    {
        double coeff_mat[] = {0, 0, -1, 0, 0, 0, -1, -2, -1, 0, -1, -2, 16, -2, -1, 0, -1, -2, -1, 0, 0, 0, -1, 0, 0};
        int size = 5;
        int middle = (size-1)/2;

        int sum = 0;

        for(int s = 0; s < size*size; s++){
            int line = s / size;
            int offset = s % size;
            int r_offset = offset - middle;
            int r_line = line - middle;

            int rx = x + r_offset;
            int ry = y + r_line;

            sum += data[(ry * width) + rx] * (coeff_mat[s]*strength);
        }

        result = sum > 255 ? 255 : (sum < 0 ? 0 : sum);
    }else{
        result = 0;
    }

    out[i] = result;
} 

int main()
{

    unsigned int image;

    ilInit();

    ilGenImages(1, &image);
    ilBindImage(image);
    ilLoadImage("in.jpg");

    int width, height, bpp, format;

    width = ilGetInteger(IL_IMAGE_WIDTH);
    height = ilGetInteger(IL_IMAGE_HEIGHT);
    bpp = ilGetInteger(IL_IMAGE_BYTES_PER_PIXEL);
    format = ilGetInteger(IL_IMAGE_FORMAT);

    // Récupération des données de l'image
    unsigned char *data = ilGetData();

    // Traitement de l'image
    unsigned char *out_grey = new unsigned char[width * height];
    unsigned char *out_blur = new unsigned char[width * height];

    // CUDA
    unsigned char* c_data;
    unsigned char* c_out;

    // Gestion de la mesure du temps
    hipEvent_t start, stop;
    float elapsedTime;

    // Gestion des erreurs CUDA
    hipError_t cudaStatus;
    hipError_t kernelStatus;

    // Creation des events pour mesure le temps
    hipEventCreate(&start);
    hipEventCreate(&stop);

    cudaStatus = hipMalloc(&c_data, 3 * width * height);
    if(cudaStatus != hipSuccess){
        std::cout << "Erreur hipMalloc c_data" << std::endl;
    }

    cudaStatus = hipMalloc(&c_out, width * height);
    if(cudaStatus != hipSuccess){
        std::cout << "Erreur hipMalloc c_out" << std::endl;
    }

    cudaStatus = hipMemcpy(c_data, data, 3 * width * height * sizeof(unsigned char), hipMemcpyHostToDevice);
    if(cudaStatus != hipSuccess){
        std::cout << "Erreur hipMemcpy c_data - HostToDevice" << std::endl;
    }

    dim3 blockDimension (32, 32);
    dim3 gridDimensions ((width/blockDimension.x)+1, (height/blockDimension.y)+1);

    // Debut du record de l'event start
    hipEventRecord(start,0);

    grayscale<<<gridDimensions, blockDimension, blockDimension.x * blockDimension.y>>>(c_data, c_out, height, width);
    kernelStatus = hipGetLastError();
    if(kernelStatus != hipSuccess){
        std::cout << "Erreur CUDA " << hipGetErrorString(kernelStatus) << std::endl;
    }
    
    cudaStatus = hipMemcpy(c_data, c_out, width * height, hipMemcpyDeviceToDevice);
    if(cudaStatus != hipSuccess){
        std::cout << "Erreur hipMemcpy c_data - DeviceToHost" << std::endl;
    }
    
    edges<<<gridDimensions, blockDimension, blockDimension.x * blockDimension.y>>>(c_data, c_out, height, width, 1.2f);
    kernelStatus = hipGetLastError();
    if(kernelStatus != hipSuccess){
        std::cout << "Erreur CUDA " << hipGetErrorString(kernelStatus) << std::endl;
    }

    cudaStatus = hipMemcpy(out_blur, c_out, width*height*sizeof(unsigned char), hipMemcpyDeviceToHost);
    if(cudaStatus != hipSuccess){
        std::cout << "Erreur hipMemcpy out_blur - DeviceToHost" << std::endl;
    }

    // Récupération du temps d'éxécution
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Time (ms) : " << elapsedTime << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(c_data);
    hipFree(c_out);

    // Placement des données dans l'image
    ilTexImage(width, height, 1, 1, IL_LUMINANCE, IL_UNSIGNED_BYTE, out_blur);

    ilEnable(IL_FILE_OVERWRITE);

    ilSaveImage("out.jpg");

    ilDeleteImages(1, &image);

    delete[] out_blur;
    delete[] out_grey;
}