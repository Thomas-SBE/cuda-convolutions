#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cstring>
#include <IL/il.h>

#define BLOCK_SIZE 32
#define CONVOLUTION_SIZE 7
#define GHOSTS_SIZE 3
#define SHARESIZE (BLOCK_SIZE+2*GHOSTS_SIZE)


__global__ void grayscale(unsigned char* data, unsigned char* out, int height, int width){
    auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
    auto y = (blockIdx.y * blockDim.y) + threadIdx.y;
    if(x >= width || y >= height) return;
    int i = (y*width)+x;
    int z = (height - y - 1) * width + x;
    out[z] = (307 * data[3 * i] + 604 * data[3 * i + 1] + 113 * data[3 * i + 2]) >> 10;
}

__global__ void edges(unsigned char* data, unsigned char* out, int height, int width, double strength){
    
    extern __shared__ unsigned char mem[];

    auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
    auto y = (blockIdx.y * blockDim.y) + threadIdx.y;
    if(x >= width || y >= height) return;
    int i = (y*width)+x;

    auto u = (blockIdx.x * (blockDim.x-GHOSTS_SIZE*2)) + threadIdx.x;
    auto v = (blockIdx.y * (blockDim.y-GHOSTS_SIZE*2)) + threadIdx.y;

    if(u < width && v < height) mem[threadIdx.y * blockDim.x + threadIdx.x] = data[v * width + u];

    __syncthreads();

    unsigned char result = mem[threadIdx.y * blockDim.x + threadIdx.x];

    if(x > 0 && x < width-1 && y > 0 && y < height-1)
    {
        double coeff_mat[] = {0, 0, 0, 5, 0, 0, 0,
                       0, 5,18,32,18, 5, 0,
                       0,18,64,100,64,18,0,
                       5,32,100,100,100,32,5,
                       0,18,64,100,64,18,0,
                       0, 5,18,32,18, 5, 0,
                       0, 0, 0, 5, 0, 0, 0};
        int size = 7;
        int middle = (size-1)/2;

        int norm_sum = 0;
        for (int i = 0; i < size*size; i++) {
        norm_sum += coeff_mat[i];
        }

        for (int i = 0; i < size*size; i++) {
        coeff_mat[i] /= norm_sum;
        }

        int sum = 0;

        for(int s = 0; s < size*size; s++){
            int line = s / size;
            int offset = s % size;
            int r_offset = offset - middle;
            int r_line = line - middle;

            int rx = threadIdx.x + r_offset;
            int ry = threadIdx.y + r_line;

            unsigned char v = mem[((ry+1)*(BLOCK_SIZE+(GHOSTS_SIZE*2)))+rx+1];

            sum += v * (coeff_mat[s]*strength);
        }

        result = sum > 255 ? 255 : (sum < 0 ? 0 : sum);
    }else{
        result = 0;
    }

    __syncthreads();

    out[i] = result;
} 

int main()
{

    unsigned int image;

    ilInit();

    ilGenImages(1, &image);
    ilBindImage(image);
    ilLoadImage("in.jpg");

    int width, height, bpp, format;

    width = ilGetInteger(IL_IMAGE_WIDTH);
    height = ilGetInteger(IL_IMAGE_HEIGHT);
    bpp = ilGetInteger(IL_IMAGE_BYTES_PER_PIXEL);
    format = ilGetInteger(IL_IMAGE_FORMAT);

    // Récupération des données de l'image
    unsigned char *data = ilGetData();

    // Traitement de l'image
    unsigned char *out_grey = new unsigned char[width * height];
    unsigned char *out_blur = new unsigned char[width * height];

    // CUDA
    unsigned char* c_data;
    unsigned char* c_out;

    // Gestion de la mesure du temps
    hipEvent_t start, stop;
    float elapsedTime;

    // Gestion des erreurs CUDA
    hipError_t cudaStatus;
    hipError_t kernelStatus;

    // Creation des events pour mesure le temps
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Debut du record de l'event start
    hipEventRecord(start,0);

    cudaStatus = hipMalloc(&c_data, 3 * width * height);
    if(cudaStatus != hipSuccess){
        std::cout << "Erreur hipMalloc c_data" << std::endl;
    }

    cudaStatus = hipMalloc(&c_out, width * height * sizeof(unsigned char));
    if(cudaStatus != hipSuccess){
        std::cout << "Erreur hipMalloc c_out" << std::endl;
    }

    cudaStatus = hipMemcpy(c_data, data, 3 * width * height * sizeof(unsigned char), hipMemcpyHostToDevice);
    if(cudaStatus != hipSuccess){
        std::cout << "Erreur hipMemcpy c_data - HostToDevice" << std::endl;
    }

    dim3 blockDimension (BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDimensions ((width/blockDimension.x)+1, (height/blockDimension.y)+1);

    grayscale<<<gridDimensions, blockDimension>>>(c_data, c_out, height, width);
    kernelStatus = hipGetLastError();
    if(kernelStatus != hipSuccess){
        std::cout << "Erreur CUDA " << hipGetErrorString(kernelStatus) << std::endl;
    }
    
    cudaStatus = hipMemcpy(c_data, c_out, width * height, hipMemcpyDeviceToDevice);
    if(cudaStatus != hipSuccess){
        std::cout << "Erreur hipMemcpy c_data - DeviceToHost" << std::endl;
    }

    edges<<<gridDimensions, blockDimension>>>(c_data, c_out, height, width, 1.2f);
    kernelStatus = hipGetLastError();
    if(kernelStatus != hipSuccess){
        std::cout << "Erreur CUDA " << hipGetErrorString(kernelStatus) << std::endl;
    }

    cudaStatus = hipMemcpy(out_blur, c_out, width*height*sizeof(unsigned char), hipMemcpyDeviceToHost);
    if(cudaStatus != hipSuccess){
        std::cout << "Erreur hipMemcpy out_blur - DeviceToHost " << hipGetErrorString(cudaStatus) << std::endl;
    }

    // Récupération du temps d'éxécution
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Time (ms) : " << elapsedTime << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(c_data);
    hipFree(c_out);

    // Placement des données dans l'image
    ilTexImage(width, height, 1, 1, IL_LUMINANCE, IL_UNSIGNED_BYTE, out_blur);

    ilEnable(IL_FILE_OVERWRITE);

    ilSaveImage("out.jpg");

    ilDeleteImages(1, &image);

    delete[] out_blur;
    delete[] out_grey;
}