#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cstring>
#include <IL/il.h>

#define BLOCK_SIZE 8
#define CONVOLUTION_SIZE 5
#define GHOSTS_SIZE 2

__global__ void grayscale(unsigned char* data, unsigned char* out, int height, int width){
    auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
    auto y = (blockIdx.y * blockDim.y) + threadIdx.y;
    if(x >= width || y >= height) return;
    int i = (y*width)+x;
    int z = (height - y - 1) * width + x;
    out[z] = (307 * data[3 * i] + 604 * data[3 * i + 1] + 113 * data[3 * i + 2]) >> 10;
}

__global__ void edges(unsigned char* data, unsigned char* out, int height, int width, double strength){
    
    __shared__ unsigned char mem[(BLOCK_SIZE+(GHOSTS_SIZE*2))*(BLOCK_SIZE+(GHOSTS_SIZE*2))];

    auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
    auto y = (blockIdx.y * blockDim.y) + threadIdx.y;
    if(x >= width || y >= height) return;
    int i = (y*width)+x;

    if(threadIdx.x == 0){
        if(threadIdx.y == 0)
            for(int a = 0; a < BLOCK_SIZE+(GHOSTS_SIZE*2); a++) mem[a] = data[i-width-1+a];
        for(int a = 0; a < BLOCK_SIZE+(GHOSTS_SIZE*2); a++) mem[(threadIdx.y+1)*(BLOCK_SIZE+(GHOSTS_SIZE*2))+a] = data[i-1+a];
        if(threadIdx.y == blockDim.y-1)
            for(int a = 0; a < BLOCK_SIZE+(GHOSTS_SIZE*2); a++) mem[(threadIdx.y+2)*(BLOCK_SIZE+(GHOSTS_SIZE*2))+a] = data[i-1+width+a];
    }

    __syncthreads();

    int pos = (threadIdx.y*(BLOCK_SIZE+(GHOSTS_SIZE*2))) + threadIdx.x + 1;

    unsigned char result = mem[pos];

    if(x > 0 && x < width-1 && y > 0 && y < height-1)
    {
        double coeff_mat[] = {0, 0, -1, 0, 0, 0, -1, -2, -1, 0, -1, -2, 16, -2, -1, 0, -1, -2, -1, 0, 0, 0, -1, 0, 0};
        int size = 5;
        int middle = (size-1)/2;

        int sum = 0;

        for(int s = 0; s < size*size; s++){
            int line = s / size;
            int offset = s % size;
            int r_offset = offset - middle;
            int r_line = line - middle;

            int rx = threadIdx.x + r_offset;
            int ry = threadIdx.y + r_line;

            unsigned char v = mem[((ry+1)*(BLOCK_SIZE+(GHOSTS_SIZE*2)))+rx+1];

            sum += v * (coeff_mat[s]*strength);
        }

        result = sum > 255 ? 255 : (sum < 0 ? 0 : sum);
    }else{
        result = 0;
    }

    out[i] = result;
} 

int main()
{

    unsigned int image;

    ilInit();

    ilGenImages(1, &image);
    ilBindImage(image);
    ilLoadImage("in.jpg");

    int width, height, bpp, format;

    width = ilGetInteger(IL_IMAGE_WIDTH);
    height = ilGetInteger(IL_IMAGE_HEIGHT);
    bpp = ilGetInteger(IL_IMAGE_BYTES_PER_PIXEL);
    format = ilGetInteger(IL_IMAGE_FORMAT);

    // Récupération des données de l'image
    unsigned char *data = ilGetData();

    // Traitement de l'image
    unsigned char *out_grey = new unsigned char[width * height];
    unsigned char *out_blur = new unsigned char[width * height];

    // CUDA
    unsigned char* c_data;
    unsigned char* c_out;

    // Gestion de la mesure du temps
    hipEvent_t start, stop;
    float elapsedTime;

    // Gestion des erreurs CUDA
    hipError_t cudaStatus;
    hipError_t kernelStatus;

    // Creation des events pour mesure le temps
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Debut du record de l'event start
    hipEventRecord(start,0);

    cudaStatus = hipMalloc(&c_data, 3 * width * height);
    if(cudaStatus != hipSuccess){
        std::cout << "Erreur hipMalloc c_data" << std::endl;
    }

    cudaStatus = hipMalloc(&c_out, width * height);
    if(cudaStatus != hipSuccess){
        std::cout << "Erreur hipMalloc c_out" << std::endl;
    }

    cudaStatus = hipMemcpy(c_data, data, 3 * width * height * sizeof(unsigned char), hipMemcpyHostToDevice);
    if(cudaStatus != hipSuccess){
        std::cout << "Erreur hipMemcpy c_data - HostToDevice" << std::endl;
    }

    dim3 blockDimension (BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDimensions ((width/blockDimension.x)+1, (height/blockDimension.y)+1);

    grayscale<<<gridDimensions, blockDimension, 0>>>(c_data, c_out, height, width);
    kernelStatus = hipGetLastError();
    if(kernelStatus != hipSuccess){
        std::cout << "Erreur CUDA " << hipGetErrorString(kernelStatus) << std::endl;
    }
    
    cudaStatus = hipMemcpy(c_data, c_out, width * height, hipMemcpyDeviceToDevice);
    if(cudaStatus != hipSuccess){
        std::cout << "Erreur hipMemcpy c_data - DeviceToHost" << std::endl;
    }

    edges<<<gridDimensions, blockDimension, 0>>>(c_data, c_out, height, width, 1.2f);
    kernelStatus = hipGetLastError();
    if(kernelStatus != hipSuccess){
        std::cout << "Erreur CUDA " << hipGetErrorString(kernelStatus) << std::endl;
    }

    cudaStatus = hipMemcpy(out_blur, c_out, width*height*sizeof(unsigned char), hipMemcpyDeviceToHost);
    if(cudaStatus != hipSuccess){
        std::cout << "Erreur hipMemcpy out_blur - DeviceToHost" << std::endl;
    }

    // Récupération du temps d'éxécution
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Time (ms) : " << elapsedTime << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(c_data);
    hipFree(c_out);

    // Placement des données dans l'image
    ilTexImage(width, height, 1, 1, IL_LUMINANCE, IL_UNSIGNED_BYTE, out_blur);

    ilEnable(IL_FILE_OVERWRITE);

    ilSaveImage("out.jpg");

    ilDeleteImages(1, &image);

    delete[] out_blur;
    delete[] out_grey;
}