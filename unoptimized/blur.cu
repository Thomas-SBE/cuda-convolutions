#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cstring>
#include <IL/il.h>

__global__ void grayscale(unsigned char* data, unsigned char* out, int height, int width){
    auto i = (blockIdx.x * blockDim.x) + threadIdx.x;
    out[i] = (307 * data[3 * i] + 604 * data[3 * i + 1] + 113 * data[3 * i + 2]) >> 10;

    __syncthreads();

    if(blockIdx.x > height/2){
        char s = out[i];
        int z = (height - blockIdx.x - 1) * blockDim.x + threadIdx.x;
        out[i] = out[z];
        out[z] = s;
    }
}

__global__ void edges(unsigned char* data, unsigned char* out, int height, int width, double strength){
    auto i = (blockIdx.x * blockDim.x) + threadIdx.x;

    unsigned char result = data[i];

    if(threadIdx.x > 0 && threadIdx.x < width-1 && blockIdx.x > 0 && blockIdx.x < height-1)
    {
        double coeff_mat[] = {0, 0, 0, 5, 0, 0, 0,
                       0, 5,18,32,18, 5, 0,
                       0,18,64,100,64,18,0,
                       5,32,100,100,100,32,5,
                       0,18,64,100,64,18,0,
                       0, 5,18,32,18, 5, 0,
                       0, 0, 0, 5, 0, 0, 0};
        int size = 7;
        int middle = (size-1)/2;

        int norm_sum = 0;
        for (int i = 0; i < size*size; i++) {
        norm_sum += coeff_mat[i];
        }

        for (int i = 0; i < size*size; i++) {
        coeff_mat[i] /= norm_sum;
        }

        int x = threadIdx.x;
        int y = blockIdx.x;
        int sum = 0;

        for(int s = 0; s < size*size; s++){
            int line = s / size;
            int offset = s % size;
            int r_offset = offset - middle;
            int r_line = line - middle;

            int rx = x + r_offset;
            int ry = y + r_line;

            sum += data[(ry * width) + rx] * (coeff_mat[s]*strength);
        }

        result = sum > 255 ? 255 : (sum < 0 ? 0 : sum);
    }else{
        result = 0;
    }

    out[i] = result;
} 

int main()
{

    unsigned int image;

    ilInit();

    ilGenImages(1, &image);
    ilBindImage(image);
    ilLoadImage("in.jpg");

    int width, height, bpp, format;

    width = ilGetInteger(IL_IMAGE_WIDTH);
    height = ilGetInteger(IL_IMAGE_HEIGHT);
    bpp = ilGetInteger(IL_IMAGE_BYTES_PER_PIXEL);
    format = ilGetInteger(IL_IMAGE_FORMAT);

    // Récupération des données de l'image
    unsigned char *data = ilGetData();

    // Traitement de l'image
    unsigned char *out_grey = new unsigned char[width * height];
    unsigned char *out_blur = new unsigned char[width * height];
 
    // Gestion de la mesure du temps
    hipEvent_t start, stop;
    float elapsedTime;

    // CUDA
    unsigned char* c_data;
    unsigned char* c_out;

    // Creation des events pour mesure le temps
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Debut du record de l'event start
    hipEventRecord(start,0);

    hipMalloc(&c_data, 3 * width * height);
    hipMalloc(&c_out, width * height);

    hipMemcpy(c_data, data, 3 * width * height * sizeof(unsigned char), hipMemcpyHostToDevice);

    grayscale<<<height, width>>>(c_data, c_out, height, width);

    hipMemcpy(c_data, c_out, width * height, hipMemcpyDeviceToDevice);

    edges<<<height, width>>>(c_data, c_out, height, width, 1.2f);

    hipMemcpy(out_blur, c_out, width*height*sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Récupération du temps d'éxécution
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Time (ms) : " << elapsedTime << std::endl;

    hipFree(c_data);
    hipFree(c_out);

    // Placement des données dans l'image
    ilTexImage(width, height, 1, 1, IL_LUMINANCE, IL_UNSIGNED_BYTE, out_blur);

    ilEnable(IL_FILE_OVERWRITE);

    ilSaveImage("out.jpg");

    ilDeleteImages(1, &image);

    delete[] out_blur;
    delete[] out_grey;
}